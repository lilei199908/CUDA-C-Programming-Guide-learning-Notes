
#include <hip/hip_runtime.h>
__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}